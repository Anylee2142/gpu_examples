// cxl_dax_gpu_demo_pageable.cu
// Build: nvcc -O3 -std=c++17 cxl_dax_gpu_demo_pageable.cu -o dax_demo_pageable

#include <fcntl.h>
#include <sys/mman.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <cstdint>
#include <iostream>

#define CUDA(call)                                                         \
    do {                                                                   \
        hipError_t _e = (call);                                           \
        if (_e != hipSuccess) {                                           \
            std::cerr << "CUDA error " << hipGetErrorString(_e)           \
                      << " at " << __FILE__ << ':' << __LINE__ << '\n';    \
            std::exit(EXIT_FAILURE);                                       \
        }                                                                  \
    } while (0)

/* ---------------- kernels ---------------- */
__global__ void write_pattern(uint32_t* buf, size_t n, uint32_t seed)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) buf[i] = seed + i;
}

__global__ void check_pattern(const uint32_t* buf, size_t n,
                              uint32_t seed, int* err)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n && buf[i] != seed + i) atomicAdd(err, 1);
}

/* ---------------- host program ---------------- */
int main()
{
    constexpr const char* DAX = "/dev/dax0.0";
    constexpr size_t BYTES  = 64ULL << 20;              // 64 MiB
    constexpr size_t WORDS  = BYTES / sizeof(uint32_t);
    constexpr uint32_t SEED = 0xDEADBEEF;

    /* 1. mmap the CXL slice (pageable) */
    int fd = open(DAX, O_RDWR | O_SYNC);
    if (fd < 0) { perror("open /dev/dax"); return EXIT_FAILURE; }

    void* h_dax = mmap(nullptr, BYTES,
                       PROT_READ | PROT_WRITE,
                       MAP_SHARED | MAP_SYNC, fd, 0);
    if (h_dax == MAP_FAILED) { perror("mmap"); return EXIT_FAILURE; }

    /* 2. Allocate device buffer */
    uint32_t* d_buf = nullptr;
    CUDA(hipMalloc(&d_buf, BYTES));

    /* 3. GPU writes pattern into *device* buffer */
    dim3 blk(256), grid((WORDS + blk.x - 1) / blk.x);
    write_pattern<<<grid, blk>>>(d_buf, WORDS, SEED);
    CUDA(hipDeviceSynchronize());

    /* 4. Copy device → CXL (pageable host) */
    CUDA(hipMemcpy(h_dax, d_buf, BYTES, hipMemcpyDeviceToHost));

    /* 5. Copy back CXL → device to verify round-trip */
    CUDA(hipMemcpy(d_buf, h_dax, BYTES, hipMemcpyHostToDevice));
    int* d_err = nullptr;
    CUDA(hipMalloc(&d_err, sizeof(int)));
    CUDA(hipMemset(d_err, 0, sizeof(int)));

    check_pattern<<<grid, blk>>>(d_buf, WORDS, SEED, d_err);
    CUDA(hipDeviceSynchronize());

    int h_err = 0;
    CUDA(hipMemcpy(&h_err, d_err, sizeof(int), hipMemcpyDeviceToHost));
    std::cout << (h_err ? "Mismatch!\n" : "Pattern verified.\n");

    /* 6. Cleanup */
    CUDA(hipFree(d_buf));
    CUDA(hipFree(d_err));
    munmap(h_dax, BYTES);
    close(fd);
    CUDA(hipDeviceReset());
    return h_err ? EXIT_FAILURE : EXIT_SUCCESS;
}
