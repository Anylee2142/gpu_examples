#include <iostream>
#include <hip/hip_runtime.h>

// CUDA kernel for vector addition
__global__ void vectorAdd(float *a, float *b, float *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    int n = 1024; // Size of the vectors
    size_t size = n * sizeof(float); // Size in bytes

    // Host memory allocation
    float *h_a = new float[n];
    float *h_b = new float[n];
    float *h_c = new float[n];

    // Initialize host vectors
    for (int i = 0; i < n; ++i) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    // Device memory allocation
    float *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copy data from host to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Configure grid and block dimensions
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);

    // Copy results from device to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Verify the results
    for (int i = 0; i < 10; ++i) {
        std::cout << h_a[i] << " + " << h_b[i] << " = " << h_c[i] << std::endl;
    }

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;

    return 0;
}
