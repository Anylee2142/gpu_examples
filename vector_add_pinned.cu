#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vecAdd(const float *a, const float *b, float *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) c[i] = a[i] + b[i];
}

int main() {
    const int N = 1 << 20;
    const size_t bytes = N * sizeof(float);

    // 1. Page-locked host allocations (pinning).
    float *h_a, *h_b, *h_c;
    hipHostAlloc(&h_a, bytes, hipHostMallocDefault);
    hipHostAlloc(&h_b, bytes, hipHostMallocDefault);
    hipHostAlloc(&h_c, bytes, hipHostMallocDefault);

    for (int i = 0; i < N; ++i) { h_a[i] = i; h_b[i] = 2*i; }

    // 2. Device buffers.
    float *d_a, *d_b, *d_c;
    hipMallocAsync(&d_a, bytes, 0);   // 0 = default stream
    hipMallocAsync(&d_b, bytes, 0);
    hipMallocAsync(&d_c, bytes, 0);

    // 3. Create a non-default stream so we can overlap.
    hipStream_t s;
    hipStreamCreate(&s);

    // 4. Async copies – these return immediately.
    hipMemcpyAsync(d_a, h_a, bytes, hipMemcpyHostToDevice, s);
    hipMemcpyAsync(d_b, h_b, bytes, hipMemcpyHostToDevice, s);

    // 5. Kernel launch in the same stream (runs after copies).
    int threads = 256;
    int blocks  = (N + threads - 1) / threads;
    vecAdd<<<blocks, threads, 0, s>>>(d_a, d_b, d_c, N);

    // 6. Copy result back, still in stream `s`.
    hipMemcpyAsync(h_c, d_c, bytes, hipMemcpyDeviceToHost, s);

    // 7. Wait for all stream work to finish.
    hipStreamSynchronize(s);

    std::cout << "c[123] = " << h_c[123] << " (should be 3×123)\n";

    // 8. Cleanup.
    hipFreeAsync(d_a, 0); hipFreeAsync(d_b, 0); hipFreeAsync(d_c, 0);
    hipHostFree(h_a); hipHostFree(h_b); hipHostFree(h_c);
    hipStreamDestroy(s);
}

